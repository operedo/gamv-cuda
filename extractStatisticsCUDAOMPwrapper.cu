#include "hip/hip_runtime.h"

/*
Add description and legal texts
*/

#ifdef _OPENMP
#include <omp.h>
#endif
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define DT double
#define MAX(x,y)  ((x) >= (y) ? (x) : (y))
#define MIN(x,y)  ((x) < (y) ? (x) : (y))
#define MEM_OPTIMIZED 0
#define THREADSX 32
#define THREADSY THREADSX

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }


#endif

void Check_CUDA_Error(const char *message)
{
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
		exit(-1);
	}
}


__device__ void computeVariogram(int i, int  j,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos){

//    int half_nd = nd/2;
//    float dx,dy,dz;

    float dx,dy,dz,dxs,dys,dzs,hs;
    int id,ii,il,it,iv;
    int lagbeg,lagend,ilag;
    float band,dcazm,dcdec,dxy,vrh,vrhpr,vrt,vrtpr,h;
    int omni;

    dx  = d_x[i] - d_x[j];
    dy  = d_y[i] - d_y[j];
    dz  = d_z[i] - d_z[j];
    dxs = dx*dx;
    dys = dy*dy;
    dzs = dz*dz;
    hs  = dxs + dys + dzs;

    if(hs <= dismxs)
    {
        if(hs < 0.0) hs = 0.0;
        h   = sqrtf(hs);


    //
    // Determine which lag this is and skip if outside the defined distance
    // tolerance:
    //
        if(h<=EPSLON){
            lagbeg = 1;
            lagend = 1;
        }
        else{
            lagbeg = -1;
            lagend = -1;
            for(ilag=2;ilag<=nlag+2;ilag++){
                if(h>=(xlag*(float)(ilag-2)-xltol) && h<=(xlag*(float)(ilag-2)+xltol)){
                    if(lagbeg<0) lagbeg = ilag;
                    lagend = ilag;
                }
            }

        }
        if(lagend>=0)
        {
        //			printf("dx=%f dy=%f dz=%fh=%f lagbeg=%d lagend=%d\n",dx,dy,dz,h,lagbeg,lagend);


        //
        // Definition of the direction corresponding to the current pair. All
        // directions are considered (overlapping of direction tolerance cones
        // is allowed):
        //


            for(id=0;id<ndir;id++){
            //
            // Check for an acceptable azimuth angle:
            //
                dxy = sqrtf(MAX((dxs+dys),0.0));
                if(dxy<EPSLON){
                    dcazm = 1.0;
                }
                else{
                    dcazm = (dx*d_uvxazm[id]+dy*d_uvyazm[id])/dxy;
                }
                if(fabsf(dcazm)>=d_csatol[id])
                {
            //
            // Check the horizontal bandwidth criteria (maximum deviation
            // perpendicular to the specified direction azimuth):
            //
                    band = d_uvxazm[id]*dy - d_uvyazm[id]*dx;
                    if(fabsf(band)<d_bandwh[id])
                    {
                        //fprintf(stdout,"dxy=%f\tdcazm=%f\tband=%f\n",dxy,dcazm,band);


                //
                // Check for an acceptable dip angle:
                //
                        if(dcazm<0.0) dxy = -dxy;
                        if(lagbeg==1)
                            dcdec = 0.0;
                        else{
                            dcdec = (dxy*d_uvhdec[id]+dz*d_uvzdec[id])/h;

                        }
                        band = d_uvhdec[id]*dz - d_uvzdec[id]*dxy;
                        if(fabsf(dcdec)>=d_csdtol[id] && fabsf(band)<=d_bandwd[id])
                        {
                    //
                    // Check the vertical bandwidth criteria (maximum deviation perpendicular
                    // to the specified dip direction):
                    //

                        //
                        // Check whether or not an omni-directional variogram is being computed:
                        //
                                omni = 0;
                                if(d_atol[id]>=90.0) omni = 1;
                        //
                        // This direction is acceptable - go ahead and compute all variograms:
                        //

                            //printf("dxy=%f dcazm=%f uvxazm[0]=%f uvyazm[0]=%f band=%f dcdec=%f omni=%d csdtol[0]=%f\n",dxy,dcazm,uvxazm[0],uvyazm[0],band,dcdec,omni,csdtol[0]);

                        //				fprintf(stdout,"dcazm=%f\tdcdec=%f\n",dcazm,dcdec);

                            for(iv=0;iv<nvarg;iv++){
                    //
                    // For this variogram, sort out which is the tail and the head value:
                    //
                                it = d_ivtype[iv];
                                if(dcazm>=0.0 && dcdec>=0.0){
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[i+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[j+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[i+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[j+ii*(maxdat)];
                                    }
                                }
                                else{
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[j+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[i+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[j+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[i+ii*(maxdat)];
                                    }
                                }
                    //
                    // Reject this pair on the basis of missing values:
                    //
                                if(vrt>=tmin && vrh>=tmin && vrt<=tmax && vrh<=tmax && it!=2 || (vrtpr>=tmin && vrhpr>=tmin && vrtpr<=tmax && vrhpr<=tmax))
                                {
                                    if(it==1 || it==5 || it>=9){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;



                                            atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrt));
                                            atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrh));
                                            atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrh-vrt)*(vrh-vrt)));

                                            if(omni){
                                                if(vrtpr>=tmin && vrhpr>=tmin && vrtpr<tmax && vrhpr<tmax){

                                                    atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                                    atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                                    atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrtpr));
                                                    atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrhpr));
                                                    atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrtpr)*(vrhpr-vrtpr)));

                                                }
                                            }
                                        }
                                    }

                                    // The Traditional Cross Semivariogram:
                //
                                    else if(it==2){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;
                                            atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(0.5*(vrt+vrtpr)));
                                            atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(0.5*(vrh+vrhpr)));
                                            atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrh)*(vrt-vrtpr)));

                                        }
                                    }
				/*

					Note: 
					If new spatial measure are requiered, they must be implemented here following the 
					previous examples, with it=1,2,5,9.

				*/
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}



__host__ void computeVariogramOMP(int i, int  j,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos, float xlaginv){

//    int half_nd = nd/2;
//    float dx,dy,dz;

    float dx,dy,dz,dxs,dys,dzs,hs;
    int id,ii,il,it,iv;
    int lagbeg,lagend,ilag;
    float band,dcazm,dcdec,dxy,vrh,vrhpr,vrt,vrtpr,h;
    int omni;

    if(i<nd && j<nd){

    dx  = d_x[i] - d_x[j];
    dy  = d_y[i] - d_y[j];
    dz  = d_z[i] - d_z[j];
    dxs = dx*dx;
    dys = dy*dy;
    dzs = dz*dz;
    hs  = dxs + dys + dzs;

    if(hs <= dismxs)
    {
        if(hs < 0.0) hs = 0.0;
        h   = sqrtf(hs);


    //
    // Determine which lag this is and skip if outside the defined distance
    // tolerance:
    //
        if(h<=EPSLON){
            lagbeg = 1;
            lagend = 1;
        }
        else{
            lagbeg = -1;
            lagend = -1;
            for(ilag=2;ilag<=nlag+2;ilag++){
                if(h>=(xlag*(float)(ilag-2)-xltol) && h<=(xlag*(float)(ilag-2)+xltol)){
                    if(lagbeg<0) lagbeg = ilag;
                    lagend = ilag;
                }
            }

            //int ilag=0;
            //int liminf=ceil((h-xltol)*xlaginv)+2;
            //int limsup=floor((h+xltol)*xlaginv)+2;
            //for(ilag=liminf;ilag<=limsup;ilag++){
            //    if(lagbeg<0)lagbeg=ilag;
            //    lagend=ilag;
            //}
	}
        if(lagend>=0)
        {
        //			printf("dx=%f dy=%f dz=%fh=%f lagbeg=%d lagend=%d\n",dx,dy,dz,h,lagbeg,lagend);


        //
        // Definition of the direction corresponding to the current pair. All
        // directions are considered (overlapping of direction tolerance cones
        // is allowed):
        //


            for(id=0;id<ndir;id++){
            //
            // Check for an acceptable azimuth angle:
            //
                dxy = sqrtf(MAX((dxs+dys),0.0));
                if(dxy<EPSLON){
                    dcazm = 1.0;
                }
                else{
                    dcazm = (dx*d_uvxazm[id]+dy*d_uvyazm[id])/dxy;
                }
                if(fabsf(dcazm)>=d_csatol[id])
                {
            //
            // Check the horizontal bandwidth criteria (maximum deviation
            // perpendicular to the specified direction azimuth):
            //
                    band = d_uvxazm[id]*dy - d_uvyazm[id]*dx;
                    if(fabsf(band)<d_bandwh[id])
                    {
                        //fprintf(stdout,"dxy=%f\tdcazm=%f\tband=%f\n",dxy,dcazm,band);


                //
                // Check for an acceptable dip angle:
                //
                        if(dcazm<0.0) dxy = -dxy;
                        if(lagbeg==1)
                            dcdec = 0.0;
                        else{
                            dcdec = (dxy*d_uvhdec[id]+dz*d_uvzdec[id])/h;

                        }
                        band = d_uvhdec[id]*dz - d_uvzdec[id]*dxy;
                        if(fabsf(dcdec)>=d_csdtol[id] && fabsf(band)<=d_bandwd[id])
                        {
                    //
                    // Check the vertical bandwidth criteria (maximum deviation perpendicular
                    // to the specified dip direction):
                    //

                        //
                        // Check whether or not an omni-directional variogram is being computed:
                        //
                                omni = 0;
                                if(d_atol[id]>=90.0) omni = 1;
                        //
                        // This direction is acceptable - go ahead and compute all variograms:
                        //

                            //printf("dxy=%f dcazm=%f uvxazm[0]=%f uvyazm[0]=%f band=%f dcdec=%f omni=%d csdtol[0]=%f\n",dxy,dcazm,uvxazm[0],uvyazm[0],band,dcdec,omni,csdtol[0]);

                        //				fprintf(stdout,"dcazm=%f\tdcdec=%f\n",dcazm,dcdec);

                            for(iv=0;iv<nvarg;iv++){
                    //
                    // For this variogram, sort out which is the tail and the head value:
                    //
                                it = d_ivtype[iv];
                                if(dcazm>=0.0 && dcdec>=0.0){
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[i+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[j+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[i+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[j+ii*(maxdat)];
                                    }
                                }
                                else{
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[j+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[i+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[j+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[i+ii*(maxdat)];
                                    }
                                }
                    //
                    // Reject this pair on the basis of missing values:
                    //
                                if(vrt>=tmin && vrh>=tmin && vrt<=tmax && vrh<=tmax && it!=2 || (vrtpr>=tmin && vrhpr>=tmin && vrtpr<=tmax && vrhpr<=tmax))
                                {
                                    if(it==1 || it==5 || it>=9){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;
						sh_np[ii + mxdlv*sh_pos]+=1.0;
						sh_dis[ii + mxdlv*sh_pos]+=(h);
						sh_tm[ii + mxdlv*sh_pos]+=(vrt);
						sh_hm[ii + mxdlv*sh_pos]+=(vrh);
						sh_gam[ii + mxdlv*sh_pos]+=((vrh-vrt)*(vrh-vrt));



                                            //atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            //atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            //atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrt));
                                            //atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrh));
                                            //atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrh-vrt)*(vrh-vrt)));

                                            if(omni){
                                                if(vrtpr>=tmin && vrhpr>=tmin && vrtpr<tmax && vrhpr<tmax){

						sh_np[ii + mxdlv*sh_pos]+=1.0;
						sh_dis[ii + mxdlv*sh_pos]+=(h);
						sh_tm[ii + mxdlv*sh_pos]+=(vrtpr);
						sh_hm[ii + mxdlv*sh_pos]+=(vrhpr);
						sh_gam[ii + mxdlv*sh_pos]+=((vrhpr-vrtpr)*(vrhpr-vrtpr));



                                                    //atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                                    //atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                                    //atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrtpr));
                                                    //atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrhpr));
                                                    //atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrtpr)*(vrhpr-vrtpr)));

                                                }
                                            }
                                        }
                                    }

                                    // The Traditional Cross Semivariogram:
                //
                                    else if(it==2){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;
						sh_np[ii + mxdlv*sh_pos]+=1.0;
						sh_dis[ii + mxdlv*sh_pos]+=(h);
						sh_tm[ii + mxdlv*sh_pos]+=(0.5*(vrt+vrtpr));
						sh_hm[ii + mxdlv*sh_pos]+=(0.5*(vrh+vrhpr));
						sh_gam[ii + mxdlv*sh_pos]+=((vrhpr-vrh)*(vrt-vrtpr));


                                            //atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            //atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            //atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(0.5*(vrt+vrtpr)));
                                            //atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(0.5*(vrh+vrhpr)));
                                            //atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrh)*(vrt-vrtpr)));

                                        }
                                    }
				/*

					Note: 
					If new spatial measure are requiered, they must be implemented here following the 
					previous examples, with it=1,2,5,9.

				*/
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    }

}







__device__ void computePointsValues(int idx, int  idy,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos,int half_nd){

    int i,j;
    j = idx + half_nd;
    i = idy;
    computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,sh_pos);

    if (idx > idy){
        i = idy;
        j = idx;

        computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos);

    } else{
        if (idx == idy){
            i = idy;
            j = idy;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr,sh_pos);
        }
        i = idx + half_nd;
        j = idy + half_nd;

        computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos);
    }


}

__host__ void computePointsValuesOMP(int idx, int  idy,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos,int half_nd, float xlaginv){

    int i,j;
    j = idx + half_nd;
    i = idy;

    computeVariogramOMP(i,j,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,sh_pos,xlaginv);

    if (idx > idy){
        i = idy;
        j = idx;

        computeVariogramOMP(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos,xlaginv);

    } else{
        if (idx == idy){
            i = idy;
            j = idy;

            computeVariogramOMP(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr,sh_pos,xlaginv);
        }
        i = idx + half_nd;
        j = idy + half_nd;

        computeVariogramOMP(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos,xlaginv);
    }


}






__global__ void variogramKernelMemoryOptimized(const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *d_np, DT *d_dis, DT *d_gam, DT *d_hm, DT *d_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int chunks_sh_mem,int frac_nd, int thres_hybrid){

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int threadId = tidx + bdimx*tidy;
    int half_nd = nd/2;
    int sh_pos = tidy%chunks_sh_mem;
    int i,j;
    int num_threads = bdimx*bdimy;
    extern __shared__ float buffer[];
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[chunks_sh_mem*mxdlv];
    float *sh_gam = &buffer[2*chunks_sh_mem*mxdlv];
    float *sh_hm = &buffer[3*chunks_sh_mem*mxdlv];
    float *sh_tm = &buffer[4*chunks_sh_mem*mxdlv];
    int init_sh_mem = threadId;

    while (init_sh_mem < chunks_sh_mem*mxdlv){
        sh_np[init_sh_mem] = 0;
        sh_dis[init_sh_mem] = 0.0;
        sh_gam[init_sh_mem] = 0.0;
        sh_hm[init_sh_mem] = 0.0;
        sh_tm[init_sh_mem] = 0.0;
        init_sh_mem += num_threads;
    }

    __syncthreads();

    //if (idx < frac_nd && idy < frac_nd){
    if (idx>=thres_hybrid && idy>=thres_hybrid && idx < frac_nd && idy < frac_nd){
        for (i = idx; i < half_nd; i += frac_nd){
            for (j = idy; j < half_nd; j += frac_nd){
                computePointsValues(i,j,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,sh_pos,half_nd);
            }
        }
    }


    __syncthreads();

    if (threadId < mxdlv){
        sh_np[threadId] += sh_np[threadId + mxdlv] + sh_np[threadId + 2*mxdlv] + sh_np[threadId + 3*mxdlv];
        sh_dis[threadId] += sh_dis[threadId + mxdlv] + sh_dis[threadId + 2*mxdlv] + sh_dis[threadId + 3*mxdlv];
        sh_tm[threadId] += sh_tm[threadId + mxdlv] + sh_tm[threadId + 2*mxdlv] + sh_tm[threadId + 3*mxdlv];
        sh_hm[threadId] += sh_hm[threadId + mxdlv] + sh_hm[threadId + 2*mxdlv] + sh_hm[threadId + 3*mxdlv];
        sh_gam[threadId] += sh_gam[threadId + mxdlv] + sh_gam[threadId + 2*mxdlv] + sh_gam[threadId + 3*mxdlv];

        atomicAdd(&d_np[threadId],sh_np[threadId]);
        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
    }

}







__global__ void variogramKernel(    const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *d_np, DT *d_dis, DT *d_gam, DT *d_hm, DT *d_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int frac_nd, int thres_hybrid){

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int threadId = tidx + bdimx*tidy;
    int half_nd = nd/2;
    int i,j;
    int num_threads = bdimx*bdimy;
    extern __shared__ float buffer[];
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[mxdlv];
    float *sh_gam = &buffer[2*mxdlv];
    float *sh_hm = &buffer[3*mxdlv];
    float *sh_tm = &buffer[4*mxdlv];

    int init_sh_mem = threadId;
    while (init_sh_mem < mxdlv){
        sh_np[init_sh_mem] = 0;
        sh_dis[init_sh_mem] = 0.0;
        sh_gam[init_sh_mem] = 0.0;
        sh_hm[init_sh_mem] = 0.0;
        sh_tm[init_sh_mem] = 0.0;
        init_sh_mem += num_threads;
    }

    __syncthreads();

    //if (idx < frac_nd && idy < frac_nd){
    if (idx>=thres_hybrid && idy>=thres_hybrid && idx < frac_nd && idy < frac_nd){
        for (i = idx; i < half_nd; i += frac_nd){
            for (j = idy; j < half_nd; j += frac_nd){
                computePointsValues(i,j,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,0,half_nd);
            }
        }
    }

    __syncthreads();

    if (threadId < mxdlv){

        atomicAdd(&d_np[threadId],sh_np[threadId]);
        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
    }
}


__host__ void variogramKernelOMP(    const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *h_np, DT *h_dis, DT *h_gam, DT *h_hm, DT *h_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int frac_nd, int thres_hybrid){
    printf("Inside host kernel\n");
    //int tidx=threadIdx.x;
    //int tidy=threadIdx.y;
    //int bidx=blockIdx.x;
    //int bidy=blockIdx.y;
    //int bdimx=blockDim.x;
    //int bdimy=blockDim.y;
    //int idx = bidx*bdimx + tidx;
    //int idy = bidy*bdimy + tidy;

    float xlaginv=1.0/xlag;

    int idx=0;
    int idy=0;
    //int threadId = tidx + bdimx*tidy;
    int threadId=0;
    int half_nd = nd/2;
    int i,j,ii,jj;
    //int num_threads = bdimx*bdimy;
    int num_threads=1;
#pragma omp parallel
{
    num_threads=omp_get_num_threads();
}
    printf("num_threads=%d\n",num_threads);
    //extern __shared__ float buffer[];
    float buffer[num_threads*1*mxdlv*5] ;
    for(i=0;i<num_threads*mxdlv*5;i++)
        buffer[i]=0;
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[mxdlv*1*num_threads];
    float *sh_gam = &buffer[2*mxdlv*1*num_threads];
    float *sh_hm = &buffer[3*mxdlv*1*num_threads];
    float *sh_tm = &buffer[4*mxdlv*1*num_threads];


    int blocksx = (frac_nd + THREADSX - 1)/THREADSX;
    int blocksy = (frac_nd + THREADSY - 1)/THREADSY;

    //int init_sh_mem = threadId;
    //while (init_sh_mem < mxdlv){
    //    sh_np[init_sh_mem] = 0;
    //    sh_dis[init_sh_mem] = 0.0;
    //    sh_gam[init_sh_mem] = 0.0;
    //    sh_hm[init_sh_mem] = 0.0;
    //    sh_tm[init_sh_mem] = 0.0;
    //    init_sh_mem += num_threads;
    //}

    //__syncthreads();

    //if (idx < frac_nd && idy < frac_nd){
    //if (idx<thres_hybrid || idy<thres_hybrid){
#pragma omp parallel shared(d_x,d_y,d_z,buffer,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,d_vr)
{
    threadId=omp_get_thread_num();

    //for(idy=0;idy<blocksy;idy++){
    //for(idx=0;idx<thres_hybrid;idx++){
    #pragma omp for 
    for (jj = 0; jj < half_nd; jj += 1){
        for (ii = 0; ii < thres_hybrid*THREADSX/2; ii += 1){
            //for (i = ii; i < nd; i += half_nd){
            //    for (j = jj; j < nd; j += half_nd){
                computePointsValuesOMP(ii,jj,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,threadId,half_nd,xlaginv);
        //    }
        //}
        //    }
        //}
    }
    }

    //for(idx=thres_hybrid;idx<blocksx;idx++){
    //for(idy=0;idy<thres_hybrid;idy++){
    #pragma omp for 
    for (ii = thres_hybrid*THREADSX/2; ii < half_nd; ii += 1){
        for (jj = 0; jj < thres_hybrid*THREADSY/2; jj += 1){
            //for (i = ii; i < nd; i += half_nd){
            //    for (j = jj; j < nd; j += half_nd){
                computePointsValuesOMP(ii,jj,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,threadId,half_nd,xlaginv);
        //    }
        //}
        //    }
        //}
    }
    }
}


    //__syncthreads();

	for(threadId=0;threadId<num_threads;threadId++){
	for(ii=0;ii<mxdlv;ii++){
		h_np[ii]+=sh_np[ii+threadId*mxdlv];
		h_dis[ii]+=sh_dis[ii+threadId*mxdlv];
		h_tm[ii]+=sh_tm[ii+threadId*mxdlv];
		h_hm[ii]+=sh_hm[ii+threadId*mxdlv];
		h_gam[ii]+=sh_gam[ii+threadId*mxdlv];
	}
	}



//    if (threadId < mxdlv){
//
//        atomicAdd(&d_np[threadId],sh_np[threadId]);
//        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
//        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
//        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
//        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
//    }
}


__host__ void variogramKernelOMPOptimized(    const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *h_np, DT *h_dis, DT *h_gam, DT *h_hm, DT *h_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int frac_nd, int thres_hybrid){
    printf("Inside host kernel\n");
    //int tidx=threadIdx.x;
    //int tidy=threadIdx.y;
    //int bidx=blockIdx.x;
    //int bidy=blockIdx.y;
    //int bdimx=blockDim.x;
    //int bdimy=blockDim.y;
    //int idx = bidx*bdimx + tidx;
    //int idy = bidy*bdimy + tidy;

    float xlaginv=1.0/xlag;

    int idx=0;
    int idy=0;
    //int threadId = tidx + bdimx*tidy;
    int threadId=0;
    int half_nd = nd/2;
    int i,j,ii,jj;
    //int num_threads = bdimx*bdimy;
    int num_threads=1;
#pragma omp parallel
{
    num_threads=omp_get_num_threads();
}
    printf("num_threads=%d\n",num_threads);
    //extern __shared__ float buffer[];
    float buffer[num_threads*1*mxdlv*5] ;
    for(i=0;i<num_threads*mxdlv*5;i++)
        buffer[i]=0;
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[mxdlv*1*num_threads];
    float *sh_gam = &buffer[2*mxdlv*1*num_threads];
    float *sh_hm = &buffer[3*mxdlv*1*num_threads];
    float *sh_tm = &buffer[4*mxdlv*1*num_threads];


    int blocksx = (frac_nd + THREADSX - 1)/THREADSX;
    int blocksy = (frac_nd + THREADSY - 1)/THREADSY;

    //int init_sh_mem = threadId;
    //while (init_sh_mem < mxdlv){
    //    sh_np[init_sh_mem] = 0;
    //    sh_dis[init_sh_mem] = 0.0;
    //    sh_gam[init_sh_mem] = 0.0;
    //    sh_hm[init_sh_mem] = 0.0;
    //    sh_tm[init_sh_mem] = 0.0;
    //    init_sh_mem += num_threads;
    //}

    //__syncthreads();

    int thresTHREADSYhalf =thres_hybrid*THREADSY/2; 
    int thresTHREADSXhalf =thres_hybrid*THREADSX/2; 
    printf("thres=%d\n",thresTHREADSYhalf);
    //if (idx < frac_nd && idy < frac_nd){
    //if (idx<thres_hybrid || idy<thres_hybrid){
#pragma omp parallel shared(d_x,d_y,d_z,buffer,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,d_vr)
{
    threadId=omp_get_thread_num();

    //for(idy=0;idy<blocksy;idy++){
    //for(idx=0;idx<thres_hybrid;idx++){
    #pragma omp for schedule(guided) 
    for (idy = 0; idy < thresTHREADSYhalf ; idy++){
        for (idx = idy; idx < nd; idx++){
    computeVariogramOMP(idx,idy,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,threadId,xlaginv);
    }
    }

//    #pragma omp for collapse(2) 
//    for (idx = thresTHREADSXhalf; idx < nd; idx += 1){
//        for (idy = 0; idy < thresTHREADSYhalf; idy += 1){
//    computeVariogramOMP(idy,idx,nd,irepo,maxdat,MAXVAR,
//        d_x,d_y,d_z,
//        EPSLON,nlag,xlag,xltol,
//        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
//        dismxs,tmax,tmin,ndir,nvarg,
//        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
//        d_csatol, d_csdtol, d_bandwh, d_bandwd,
//        d_atol,
//        d_ivtype, d_ivtail, d_ivhead,
//        d_vr,threadId,xlaginv);
//    }
//    }

    #pragma omp for schedule(guided)  
    for (idx = half_nd; idx < half_nd + thresTHREADSXhalf; idx += 1){
        for (idy = thresTHREADSYhalf; idy < idx; idy += 1){
    //printf("Entro loop 2\n");
    computeVariogramOMP(idx,idy,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,threadId,xlaginv);
    }
    }

//    #pragma omp for collapse(2) 
//    for (idy = half_nd; idy < half_nd + thresTHREADSYhalf ; idy += 1){
//        for (idx = idy; idx < half_nd + thresTHREADSXhalf; idx += 1){
//    computeVariogramOMP(idy,idx,nd,irepo,maxdat,MAXVAR,
//        d_x,d_y,d_z,
//        EPSLON,nlag,xlag,xltol,
//        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
//        dismxs,tmax,tmin,ndir,nvarg,
//        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
//        d_csatol, d_csdtol, d_bandwh, d_bandwd,
//        d_atol,
//        d_ivtype, d_ivtail, d_ivhead,
//        d_vr,threadId,xlaginv);
//    }
//    }

    #pragma omp for schedule(guided)  
    for (idy = half_nd; idy < half_nd + thresTHREADSYhalf; idy += 1){
        for (idx = half_nd + thresTHREADSXhalf; idx < nd ; idx += 1){
    //printf("Entro loop 3\n");
    computeVariogramOMP(idx,idy,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,threadId,xlaginv);
    }
    }

}

    //__syncthreads();

	for(threadId=0;threadId<num_threads;threadId++){
	for(ii=0;ii<mxdlv;ii++){
		h_np[ii]+=sh_np[ii+threadId*mxdlv];
		h_dis[ii]+=sh_dis[ii+threadId*mxdlv];
		h_tm[ii]+=sh_tm[ii+threadId*mxdlv];
		h_hm[ii]+=sh_hm[ii+threadId*mxdlv];
		h_gam[ii]+=sh_gam[ii+threadId*mxdlv];
	}
	}



//    if (threadId < mxdlv){
//
//        atomicAdd(&d_np[threadId],sh_np[threadId]);
//        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
//        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
//        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
//        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
//    }
}



extern "C" int extractstatisticscudaompwrapper_(
                            //      integer nd,irepo,maxdat,MAXVAR
                                int *nd, int *irepo, int *maxdat, int *MAXVAR,
                            //      real x(maxdat),y(maxdat),z(maxdat)
                                float *x, float *y, float *z,
                            //      real EPSLON
                                float *EPSLON,
                            //      integer nlag
                                int *nlag,
                            //      real xlag,xltol
                                float *xlag, float *xltol,
                            //      integer mxdlv
                                int *mxdlv,
                            //      real*8 np(mxdlv),dis(mxdlv),gam(mxdlv),hm(mxdlv),
                            //     + tm(mxdlv),hv(mxdlv),tv(mxdlv)
                                double *np, double *dis, double *gam, double *hm, double *tm, double *hv, double *tv,
                            //      integer numThreads
                                int *numThreads,
                            //      real*8 reducedVariables(7,mxdlv,numThreads)
                                double *reducedVariables,
                            //      real dismxs,tmax,tmin
                                float *dismxs, float *tmax, float *tmin,
                            //      integer ndir,nvarg
                                int *ndir, int *nvarg,
                            //      real uvxazm(100),uvyazm(100),uvzdec(100),uvhdec(100)
                                float *uvxazm, float *uvyazm, float *uvzdec, float *uvhdec,
                            //      real csatol(100),csdtol(100),bandwh(ndir),bandwd(ndir)
                                float *csatol, float *csdtol, float *bandwh, float *bandwd,
                            //      real atol(ndir)
                                float *atol,
                            //      integer ivtype(nvarg),ivtail(nvarg),ivhead(nvarg)
                                int *ivtype, int *ivtail, int *ivhead,
                            //      real vr(maxdat,MAXVAR)
                                float *vr)
{
	float *d_x,*d_y,*d_z;
	DT *d_np,*d_dis,*d_gam,*d_hm,*d_tm;
	DT *h_np,*h_dis,*h_gam,*h_hm,*h_tm;
	DT *hh_np,*hh_dis,*hh_gam,*hh_hm,*hh_tm;
	float *d_uvxazm,*d_uvyazm,*d_uvzdec,*d_uvhdec,*d_csatol,*d_csdtol,*d_bandwh,*d_bandwd,*d_atol,*d_vr;
	int *d_ivtype,*d_ivtail,*d_ivhead;



    	hipSetDevice(0);
	hipStream_t streamid;
	hipStreamCreate(&streamid);

	// CUDA kernel will process the first half of the data.
	float thres_factor = 1.0f;
	int thres_hybrid = (int)(thres_factor*(float)(*maxdat/THREADSX));
	



    	dim3 threads(THREADSX,THREADSY,1);
	int frac_nd;

    	int chunk_sh_mem = 4;
	h_np = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_dis = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_gam = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_hm = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_tm = (DT*)malloc(sizeof(DT)* *mxdlv);
	hh_np = (DT*)malloc(sizeof(DT)* *mxdlv);
	hh_dis = (DT*)malloc(sizeof(DT)* *mxdlv);
	hh_gam = (DT*)malloc(sizeof(DT)* *mxdlv);
	hh_hm = (DT*)malloc(sizeof(DT)* *mxdlv);
	hh_tm = (DT*)malloc(sizeof(DT)* *mxdlv);


	int shared_mem_size;
    	int i;
    	for (i = 0; i < *mxdlv; i++){
        	h_np[i] = 0.0;
        	h_dis[i] = 0.0;
        	h_gam[i] = 0.0;
        	h_hm[i] = 0.0;
        	h_tm[i] = 0.0;
        	hh_np[i] = 0.0;
        	hh_dis[i] = 0.0;
        	hh_gam[i] = 0.0;
        	hh_hm[i] = 0.0;
        	hh_tm[i] = 0.0;
    	}
   	hipMalloc( (void **)&d_x, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_y, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_z, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_np, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_dis, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_gam, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_hm, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_tm, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_uvxazm, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvyazm, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvzdec, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvhdec, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_csatol, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_csdtol, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_bandwh, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_bandwd, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_atol, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_vr, sizeof(float) * (*maxdat* *MAXVAR) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_ivtype, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMalloc( (void **)&d_ivtail, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMalloc( (void **)&d_ivhead, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMemcpyAsync( d_x, x,sizeof(float) * (*maxdat), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpyAsync( d_y, y,sizeof(float) * (*maxdat), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpyAsync( d_z, z,sizeof(float) * (*maxdat), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpyAsync( d_np, h_np,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpyAsync( d_dis, h_dis,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpyAsync( d_gam, h_gam,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpyAsync( d_hm, h_hm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpyAsync( d_tm, h_tm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpyAsync( d_uvxazm, uvxazm,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_uvyazm, uvyazm,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_uvzdec, uvzdec,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_uvhdec, uvhdec,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_csatol, csatol,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_csdtol, csdtol,sizeof(float) * (100), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_bandwh, bandwh,sizeof(float) * (*ndir), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_bandwd, bandwd,sizeof(float) * (*ndir), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_atol, atol,sizeof(float) * (*ndir), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_vr, vr,sizeof(float) * (*maxdat* *MAXVAR), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpyAsync( d_ivtype, ivtype,sizeof(float) * (*nvarg), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipMemcpyAsync( d_ivtail, ivtail,sizeof(float) * (*nvarg), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipMemcpyAsync( d_ivhead, ivhead,sizeof(float) * (*nvarg), hipMemcpyHostToDevice , streamid);
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipEvent_t start, stop;
   	float time;


	//int num_devices;
	//hipGetDeviceCount(&num_devices);
	//printf("num_devices=%d\n",num_devices);



	frac_nd = *maxdat/THREADSX;
       	dim3 blocks( (frac_nd + threads.x - 1)/threads.x,(frac_nd + threads.y - 1)/threads.y,1 );
       	hipEventCreate(&start);
       	hipEventCreate(&stop);
       	hipEventRecord(start, streamid);
    	if (MEM_OPTIMIZED){
       		shared_mem_size = sizeof(DT)*(*mxdlv*5*chunk_sh_mem);
		printf("Starting asynchronous CUDA kernel (mem-opt)...\n");
        	variogramKernelMemoryOptimized<<< blocks, threads,shared_mem_size,streamid >>>(*nd,*irepo,*maxdat,*MAXVAR,
                                            d_x,d_y,d_z,
                                            *EPSLON,
                                            *nlag,
                                            *xlag,*xltol,
                                            *mxdlv,
                                            d_np,d_dis,d_gam,d_hm,d_tm,
                                            *dismxs,*tmax,*tmin,
                                            *ndir,*nvarg,
                                            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                                            d_csatol,d_csdtol,d_bandwh,d_bandwd,
                                            d_atol,
                                            d_ivtype,d_ivtail,d_ivhead,
                                            d_vr,chunk_sh_mem,frac_nd,thres_hybrid);
        	//hipDeviceSynchronize();
    	}else{
        	shared_mem_size = sizeof(DT)*(*mxdlv*5);
		printf("Starting asynchronous CUDA kernel...\n");
        	variogramKernel<<< blocks, threads,shared_mem_size, streamid >>>(*nd,*irepo,*maxdat,*MAXVAR,
                                            d_x,d_y,d_z,
                                            *EPSLON,
                                            *nlag,
                                            *xlag,*xltol,
                                            *mxdlv,
                                            d_np,d_dis,d_gam,d_hm,d_tm,
                                            *dismxs,*tmax,*tmin,
                                            *ndir,*nvarg,
                                            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                                            d_csatol,d_csdtol,d_bandwh,d_bandwd,
                                            d_atol,
                                            d_ivtype,d_ivtail,d_ivhead,
                                            d_vr,frac_nd,thres_hybrid);
	
     

		//hipDeviceSynchronize();
    	}
       	Check_CUDA_Error("fitness kernel");
       	hipEventRecord(stop, streamid);
       	hipEventSynchronize(stop);
       	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the Optimized kernel: %f ms\n", time);
       	printf ("GPU time: %f\n", time/1000);
       	printf("------------------------------\n");



	//variogramKernelOMP(*nd,*irepo,*maxdat,*MAXVAR,
	variogramKernelOMPOptimized(*nd,*irepo,*maxdat,*MAXVAR,
                                    x,y,z,
                                    *EPSLON,
                                    *nlag,
                                    *xlag,*xltol,
                                    *mxdlv,
                                    hh_np,hh_dis,hh_gam,hh_hm,hh_tm,
                                    *dismxs,*tmax,*tmin,
                                    *ndir,*nvarg,
                                    uvxazm,uvyazm,uvzdec,uvhdec,
                                    csatol,csdtol,bandwh,bandwd,
                                    atol,
                                    ivtype,ivtail,ivhead,
                                    vr,frac_nd,thres_hybrid);
 



	//hipStreamSynchronize(0);

       	hipStreamSynchronize(streamid);

    	hipMemcpyAsync( h_np, d_np,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost, streamid);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpyAsync( h_dis, d_dis,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost, streamid);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpyAsync( h_gam, d_gam,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost, streamid);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpyAsync( h_hm, d_hm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost, streamid);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpyAsync( h_tm, d_tm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost, streamid);
    	//Check_CUDA_Error("cpy d -> h");

   	// printf("np, dis, gam, hm, tm\n");
	//    float sum_np = 0.0;
 
	hipStreamDestroy(streamid);


   	for (i = 0; i < *mxdlv; i++){
        	np[i] = (double)h_np[i];
        	dis[i] = (double)h_dis[i];
        	gam[i] = (double)h_gam[i];
        	hm[i] = (double)h_hm[i];
        	tm[i] = (double)h_tm[i];
      	//  printf("%lf\t, %lf\t, %lf\t, %lf\t, %lf\n",np[i],dis[i],gam[i],hm[i],tm[i]);
    	}
   	for (i = 0; i < *mxdlv; i++){
        	np[i] += (double)hh_np[i];
        	dis[i] += (double)hh_dis[i];
        	gam[i] += (double)hh_gam[i];
        	hm[i] += (double)hh_hm[i];
        	tm[i] += (double)hh_tm[i];
      	//  printf("%lf\t, %lf\t, %lf\t, %lf\t, %lf\n",np[i],dis[i],gam[i],hm[i],tm[i]);
    	}



    	hipFree(d_x);
    	hipFree(d_y);
    	hipFree(d_z);
    	hipFree(d_np);
    	hipFree(d_dis);
    	hipFree(d_gam);
    	hipFree(d_hm);
    	hipFree(d_tm);
    	hipFree(d_uvxazm);
    	hipFree(d_uvyazm);
    	hipFree(d_uvzdec);
    	hipFree(d_uvhdec);
    	hipFree(d_csatol);
    	hipFree(d_csdtol);
    	hipFree(d_bandwh);
    	hipFree(d_bandwd);
    	hipFree(d_atol);
    	hipFree(d_vr);
    	hipFree(d_ivtype);
    	hipFree(d_ivtail);
    	hipFree(d_ivhead);
    	free(h_np);
    	free(h_dis);
    	free(h_gam);
    	free(h_hm);
    	free(h_tm);
    	free(hh_np);
    	free(hh_dis);
    	free(hh_gam);
    	free(hh_hm);
    	free(hh_tm);
	return 0;
//end routine
}







/*

TODO:

- Add documentation in each routine
- Add diagrams (UML?) of the sequence and interactions between the CPU and GPU
- Document the memory optimization proposed.

*/








