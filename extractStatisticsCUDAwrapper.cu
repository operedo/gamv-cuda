
/*
Add description and legal texts
*/


#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>



#define DT double
#define MAX(x,y)  ((x) >= (y) ? (x) : (y))
#define MIN(x,y)  ((x) < (y) ? (x) : (y))
#define MEM_OPTIMIZED 0
#define THREADSX 32
#define THREADSY THREADSX

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }


#endif

void Check_CUDA_Error(const char *message)
{
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
		exit(-1);
	}
}


__device__ void computeVariogram(int i, int  j,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos){

//    int half_nd = nd/2;
//    float dx,dy,dz;

    float dx,dy,dz,dxs,dys,dzs,hs;
    int id,ii,il,it,iv;
    int lagbeg,lagend,ilag;
    float band,dcazm,dcdec,dxy,vrh,vrhpr,vrt,vrtpr,h;
    int omni;

    dx  = d_x[i] - d_x[j];
    dy  = d_y[i] - d_y[j];
    dz  = d_z[i] - d_z[j];
    dxs = dx*dx;
    dys = dy*dy;
    dzs = dz*dz;
    hs  = dxs + dys + dzs;

    if(hs <= dismxs)
    {
        if(hs < 0.0) hs = 0.0;
        h   = sqrtf(hs);


    //
    // Determine which lag this is and skip if outside the defined distance
    // tolerance:
    //
        if(h<=EPSLON){
            lagbeg = 1;
            lagend = 1;
        }
        else{
            lagbeg = -1;
            lagend = -1;
            for(ilag=2;ilag<=nlag+2;ilag++){
                if(h>=(xlag*(float)(ilag-2)-xltol) && h<=(xlag*(float)(ilag-2)+xltol)){
                    if(lagbeg<0) lagbeg = ilag;
                    lagend = ilag;
                }
            }

        }
        if(lagend>=0)
        {
        //			printf("dx=%f dy=%f dz=%fh=%f lagbeg=%d lagend=%d\n",dx,dy,dz,h,lagbeg,lagend);


        //
        // Definition of the direction corresponding to the current pair. All
        // directions are considered (overlapping of direction tolerance cones
        // is allowed):
        //


            for(id=0;id<ndir;id++){
            //
            // Check for an acceptable azimuth angle:
            //
                dxy = sqrtf(MAX((dxs+dys),0.0));
                if(dxy<EPSLON){
                    dcazm = 1.0;
                }
                else{
                    dcazm = (dx*d_uvxazm[id]+dy*d_uvyazm[id])/dxy;
                }
                if(fabsf(dcazm)>=d_csatol[id])
                {
            //
            // Check the horizontal bandwidth criteria (maximum deviation
            // perpendicular to the specified direction azimuth):
            //
                    band = d_uvxazm[id]*dy - d_uvyazm[id]*dx;
                    if(fabsf(band)<d_bandwh[id])
                    {
                        //fprintf(stdout,"dxy=%f\tdcazm=%f\tband=%f\n",dxy,dcazm,band);


                //
                // Check for an acceptable dip angle:
                //
                        if(dcazm<0.0) dxy = -dxy;
                        if(lagbeg==1)
                            dcdec = 0.0;
                        else{
                            dcdec = (dxy*d_uvhdec[id]+dz*d_uvzdec[id])/h;

                        }
                        band = d_uvhdec[id]*dz - d_uvzdec[id]*dxy;
                        if(fabsf(dcdec)>=d_csdtol[id] && fabsf(band)<=d_bandwd[id])
                        {
                    //
                    // Check the vertical bandwidth criteria (maximum deviation perpendicular
                    // to the specified dip direction):
                    //

                        //
                        // Check whether or not an omni-directional variogram is being computed:
                        //
                                omni = 0;
                                if(d_atol[id]>=90.0) omni = 1;
                        //
                        // This direction is acceptable - go ahead and compute all variograms:
                        //

                            //printf("dxy=%f dcazm=%f uvxazm[0]=%f uvyazm[0]=%f band=%f dcdec=%f omni=%d csdtol[0]=%f\n",dxy,dcazm,uvxazm[0],uvyazm[0],band,dcdec,omni,csdtol[0]);

                        //				fprintf(stdout,"dcazm=%f\tdcdec=%f\n",dcazm,dcdec);

                            for(iv=0;iv<nvarg;iv++){
                    //
                    // For this variogram, sort out which is the tail and the head value:
                    //
                                it = d_ivtype[iv];
                                if(dcazm>=0.0 && dcdec>=0.0){
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[i+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[j+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[i+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[j+ii*(maxdat)];
                                    }
                                }
                                else{
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[j+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[i+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[j+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[i+ii*(maxdat)];
                                    }
                                }
                    //
                    // Reject this pair on the basis of missing values:
                    //
                                if(vrt>=tmin && vrh>=tmin && vrt<=tmax && vrh<=tmax && it!=2 || (vrtpr>=tmin && vrhpr>=tmin && vrtpr<=tmax && vrhpr<=tmax))
                                {
                                    if(it==1 || it==5 || it>=9){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;



                                            atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrt));
                                            atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrh));
                                            atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrh-vrt)*(vrh-vrt)));

                                            if(omni){
                                                if(vrtpr>=tmin && vrhpr>=tmin && vrtpr<tmax && vrhpr<tmax){

                                                    atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                                    atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                                    atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(vrtpr));
                                                    atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(vrhpr));
                                                    atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrtpr)*(vrhpr-vrtpr)));

                                                }
                                            }
                                        }
                                    }

                                    // The Traditional Cross Semivariogram:
                //
                                    else if(it==2){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;
                                            atomicAdd(&sh_np[ii + mxdlv*sh_pos],1.0);
                                            atomicAdd(&sh_dis[ii + mxdlv*sh_pos],(h));
                                            atomicAdd(&sh_tm[ii + mxdlv*sh_pos],(0.5*(vrt+vrtpr)));
                                            atomicAdd(&sh_hm[ii + mxdlv*sh_pos],(0.5*(vrh+vrhpr)));
                                            atomicAdd(&sh_gam[ii + mxdlv*sh_pos],((vrhpr-vrh)*(vrt-vrtpr)));

                                        }
                                    }
				/*

					Note: 
					If new spatial measure are requiered, they must be implemented here following the 
					previous examples, with it=1,2,5,9.

				*/
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}







__device__ void computePointsValues(int idx, int  idy,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr, int sh_pos,int half_nd){

    int i,j;
    j = idx + half_nd;
    i = idy;
    computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
        d_x,d_y,d_z,
        EPSLON,nlag,xlag,xltol,
        mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
        dismxs,tmax,tmin,ndir,nvarg,
        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
        d_csatol, d_csdtol, d_bandwh, d_bandwd,
        d_atol,
        d_ivtype, d_ivtail, d_ivhead,
        d_vr,sh_pos);

    if (idx > idy){
        i = idy;
        j = idx;

        computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos);

    } else{
        if (idx == idy){
            i = idy;
            j = idy;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr,sh_pos);
        }
        i = idx + half_nd;
        j = idy + half_nd;

        computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr,sh_pos);
    }


}






__global__ void variogramKernelMemoryOptimized(const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *d_np, DT *d_dis, DT *d_gam, DT *d_hm, DT *d_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int chunks_sh_mem,int frac_nd){

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int threadId = tidx + bdimx*tidy;
    int half_nd = nd/2;
    int sh_pos = tidy%chunks_sh_mem;
    int i,j;
    int num_threads = bdimx*bdimy;
    extern __shared__ float buffer[];
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[chunks_sh_mem*mxdlv];
    float *sh_gam = &buffer[2*chunks_sh_mem*mxdlv];
    float *sh_hm = &buffer[3*chunks_sh_mem*mxdlv];
    float *sh_tm = &buffer[4*chunks_sh_mem*mxdlv];
    int init_sh_mem = threadId;

    while (init_sh_mem < chunks_sh_mem*mxdlv){
        sh_np[init_sh_mem] = 0;
        sh_dis[init_sh_mem] = 0.0;
        sh_gam[init_sh_mem] = 0.0;
        sh_hm[init_sh_mem] = 0.0;
        sh_tm[init_sh_mem] = 0.0;
        init_sh_mem += num_threads;
    }

    __syncthreads();

    if (idx < frac_nd && idy < frac_nd){
        for (i = idx; i < half_nd; i += frac_nd){
            for (j = idy; j < half_nd; j += frac_nd){
                computePointsValues(i,j,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,sh_pos,half_nd);
            }
        }
    }


    __syncthreads();

    if (threadId < mxdlv){
        sh_np[threadId] += sh_np[threadId + mxdlv] + sh_np[threadId + 2*mxdlv] + sh_np[threadId + 3*mxdlv];
        sh_dis[threadId] += sh_dis[threadId + mxdlv] + sh_dis[threadId + 2*mxdlv] + sh_dis[threadId + 3*mxdlv];
        sh_tm[threadId] += sh_tm[threadId + mxdlv] + sh_tm[threadId + 2*mxdlv] + sh_tm[threadId + 3*mxdlv];
        sh_hm[threadId] += sh_hm[threadId + mxdlv] + sh_hm[threadId + 2*mxdlv] + sh_hm[threadId + 3*mxdlv];
        sh_gam[threadId] += sh_gam[threadId + mxdlv] + sh_gam[threadId + 2*mxdlv] + sh_gam[threadId + 3*mxdlv];

        atomicAdd(&d_np[threadId],sh_np[threadId]);
        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
    }

}







__global__ void variogramKernel(    const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *d_np, DT *d_dis, DT *d_gam, DT *d_hm, DT *d_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr,int frac_nd){

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int threadId = tidx + bdimx*tidy;
    int half_nd = nd/2;
    int i,j;
    int num_threads = bdimx*bdimy;
    extern __shared__ float buffer[];
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[mxdlv];
    float *sh_gam = &buffer[2*mxdlv];
    float *sh_hm = &buffer[3*mxdlv];
    float *sh_tm = &buffer[4*mxdlv];

    int init_sh_mem = threadId;
    while (init_sh_mem < mxdlv){
        sh_np[init_sh_mem] = 0;
        sh_dis[init_sh_mem] = 0.0;
        sh_gam[init_sh_mem] = 0.0;
        sh_hm[init_sh_mem] = 0.0;
        sh_tm[init_sh_mem] = 0.0;
        init_sh_mem += num_threads;
    }

    __syncthreads();

    if (idx < frac_nd && idy < frac_nd){
        for (i = idx; i < half_nd; i += frac_nd){
            for (j = idy; j < half_nd; j += frac_nd){
                computePointsValues(i,j,nd,irepo,maxdat,MAXVAR,
                    d_x,d_y,d_z,
                    EPSLON,nlag,xlag,xltol,
                    mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                    dismxs,tmax,tmin,ndir,nvarg,
                    d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                    d_csatol, d_csdtol, d_bandwh, d_bandwd,
                    d_atol,
                    d_ivtype, d_ivtail, d_ivhead,
                    d_vr,0,half_nd);
            }
        }
    }

    __syncthreads();

    if (threadId < mxdlv){

        atomicAdd(&d_np[threadId],sh_np[threadId]);
        atomicAdd(&d_dis[threadId],sh_dis[threadId]);
        atomicAdd(&d_tm[threadId],sh_tm[threadId]);
        atomicAdd(&d_hm[threadId],sh_hm[threadId]);
        atomicAdd(&d_gam[threadId],sh_gam[threadId]);
    }
}





extern "C" int extractstatisticscudawrapper_(
                            //      integer nd,irepo,maxdat,MAXVAR
                                int *nd, int *irepo, int *maxdat, int *MAXVAR,
                            //      real x(maxdat),y(maxdat),z(maxdat)
                                float *x, float *y, float *z,
                            //      real EPSLON
                                float *EPSLON,
                            //      integer nlag
                                int *nlag,
                            //      real xlag,xltol
                                float *xlag, float *xltol,
                            //      integer mxdlv
                                int *mxdlv,
                            //      real*8 np(mxdlv),dis(mxdlv),gam(mxdlv),hm(mxdlv),
                            //     + tm(mxdlv),hv(mxdlv),tv(mxdlv)
                                double *np, double *dis, double *gam, double *hm, double *tm, double *hv, double *tv,
                            //      integer numThreads
                                int *numThreads,
                            //      real*8 reducedVariables(7,mxdlv,numThreads)
                                double *reducedVariables,
                            //      real dismxs,tmax,tmin
                                float *dismxs, float *tmax, float *tmin,
                            //      integer ndir,nvarg
                                int *ndir, int *nvarg,
                            //      real uvxazm(100),uvyazm(100),uvzdec(100),uvhdec(100)
                                float *uvxazm, float *uvyazm, float *uvzdec, float *uvhdec,
                            //      real csatol(100),csdtol(100),bandwh(ndir),bandwd(ndir)
                                float *csatol, float *csdtol, float *bandwh, float *bandwd,
                            //      real atol(ndir)
                                float *atol,
                            //      integer ivtype(nvarg),ivtail(nvarg),ivhead(nvarg)
                                int *ivtype, int *ivtail, int *ivhead,
                            //      real vr(maxdat,MAXVAR)
                                float *vr)
{
	float *d_x,*d_y,*d_z;
	DT *d_np,*d_dis,*d_gam,*d_hm,*d_tm;
	DT *h_np,*h_dis,*h_gam,*h_hm,*h_tm;
	float *d_uvxazm,*d_uvyazm,*d_uvzdec,*d_uvhdec,*d_csatol,*d_csdtol,*d_bandwh,*d_bandwd,*d_atol,*d_vr;
	int *d_ivtype,*d_ivtail,*d_ivhead;
    	hipSetDevice(0);
    	dim3 threads(THREADSX,THREADSY,1);
	int frac_nd;

    	int chunk_sh_mem = 4;
	h_np = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_dis = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_gam = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_hm = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_tm = (DT*)malloc(sizeof(DT)* *mxdlv);
	int shared_mem_size;
    	int i;
    	for (i = 0; i < *mxdlv; i++){
        	h_np[i] = 0.0;
        	h_dis[i] = 0.0;
        	h_gam[i] = 0.0;
        	h_hm[i] = 0.0;
        	h_tm[i] = 0.0;
    	}
   	hipMalloc( (void **)&d_x, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_y, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_z, sizeof(float) * (*maxdat) );
   	//Check_CUDA_Error("malloc coord");
   	hipMalloc( (void **)&d_np, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_dis, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_gam, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_hm, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_tm, sizeof(DT) * (*mxdlv) );
   	//Check_CUDA_Error("malloc np, dis, gam, hm, tm");
   	hipMalloc( (void **)&d_uvxazm, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvyazm, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvzdec, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_uvhdec, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_csatol, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_csdtol, sizeof(float) * (100) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_bandwh, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_bandwd, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_atol, sizeof(float) * (*ndir) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_vr, sizeof(float) * (*maxdat* *MAXVAR) );
   	//Check_CUDA_Error("small mallocs ");
   	hipMalloc( (void **)&d_ivtype, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMalloc( (void **)&d_ivtail, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMalloc( (void **)&d_ivhead, sizeof(float) * (*nvarg) );
   	//Check_CUDA_Error("iv mallocs");
   	hipMemcpy( d_x, x,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpy( d_y, y,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpy( d_z, z,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy coords h -> d");
   	hipMemcpy( d_np, h_np,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpy( d_dis, h_dis,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpy( d_gam, h_gam,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpy( d_hm, h_hm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpy( d_tm, h_tm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
   	hipMemcpy( d_uvxazm, uvxazm,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_uvyazm, uvyazm,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_uvzdec, uvzdec,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_uvhdec, uvhdec,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_csatol, csatol,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_csdtol, csdtol,sizeof(float) * (100), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_bandwh, bandwh,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_bandwd, bandwd,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_atol, atol,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_vr, vr,sizeof(float) * (*maxdat* *MAXVAR), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy small data h -> d");
   	hipMemcpy( d_ivtype, ivtype,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipMemcpy( d_ivtail, ivtail,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipMemcpy( d_ivhead, ivhead,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
   	//Check_CUDA_Error("cpy iv var h -> d");
   	hipEvent_t start, stop;
   	float time;

    	if (MEM_OPTIMIZED){
        	printf("---------\nmaxdat %d nd %d\n---------\n",*maxdat,*nd);
        	frac_nd = *maxdat/THREADSX;
        	dim3 blocks( (frac_nd + threads.x - 1)/threads.x,(frac_nd + threads.y - 1)/threads.y,1 );
        	shared_mem_size = sizeof(DT)*(*mxdlv*5*chunk_sh_mem);
        	hipEventCreate(&start);
        	hipEventCreate(&stop);
        	hipEventRecord(start, 0);
        	variogramKernelMemoryOptimized<<< blocks, threads,shared_mem_size >>>(*nd,*irepo,*maxdat,*MAXVAR,
                                            d_x,d_y,d_z,
                                            *EPSLON,
                                            *nlag,
                                            *xlag,*xltol,
                                            *mxdlv,
                                            d_np,d_dis,d_gam,d_hm,d_tm,
                                            *dismxs,*tmax,*tmin,
                                            *ndir,*nvarg,
                                            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                                            d_csatol,d_csdtol,d_bandwh,d_bandwd,
                                            d_atol,
                                            d_ivtype,d_ivtail,d_ivhead,
                                            d_vr,chunk_sh_mem,frac_nd);
        	hipDeviceSynchronize();
        	Check_CUDA_Error("fitness kernel");
        	hipEventRecord(stop, 0);
        	hipEventSynchronize(stop);
        	hipEventElapsedTime(&time, start, stop);
                printf ("Time for the Optimized kernel: %f ms\n", time);
        	printf ("GPU time: %f\n", time/1000);
        	printf("------------------------------\n");
    	}else{
        	frac_nd = *maxdat/THREADSX;
        	dim3 blocks( (frac_nd + threads.x - 1)/threads.x,(frac_nd + threads.y - 1)/threads.y,1 );
        	shared_mem_size = sizeof(DT)*(*mxdlv*5);
        	hipEventCreate(&start);
        	hipEventCreate(&stop);
        	hipEventRecord(start, 0);
        	variogramKernel<<< blocks, threads,shared_mem_size >>>(*nd,*irepo,*maxdat,*MAXVAR,
                                            d_x,d_y,d_z,
                                            *EPSLON,
                                            *nlag,
                                            *xlag,*xltol,
                                            *mxdlv,
                                            d_np,d_dis,d_gam,d_hm,d_tm,
                                            *dismxs,*tmax,*tmin,
                                            *ndir,*nvarg,
                                            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                                            d_csatol,d_csdtol,d_bandwh,d_bandwd,
                                            d_atol,
                                            d_ivtype,d_ivtail,d_ivhead,
                                            d_vr,frac_nd);
        	hipDeviceSynchronize();
        	Check_CUDA_Error("fitness kernel");
        	hipEventRecord(stop, 0);
        	hipEventSynchronize(stop);
        	hipEventElapsedTime(&time, start, stop);
                printf ("Time for variogram kernel: %f ms\n", time);
        	printf ("GPU time: %f\n", time/1000);
        	printf("------------------------------\n");
    	}

    	hipMemcpy( h_np, d_np,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpy( h_dis, d_dis,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpy( h_gam, d_gam,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpy( h_hm, d_hm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    	//Check_CUDA_Error("cpy d -> h");
    	hipMemcpy( h_tm, d_tm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    	//Check_CUDA_Error("cpy d -> h");

   	// printf("np, dis, gam, hm, tm\n");
	//    float sum_np = 0.0;
    	for (i = 0; i < *mxdlv; i++){
        	np[i] = (double)h_np[i];
        	dis[i] = (double)h_dis[i];
        	gam[i] = (double)h_gam[i];
        	hm[i] = (double)h_hm[i];
        	tm[i] = (double)h_tm[i];
      	//  printf("%lf\t, %lf\t, %lf\t, %lf\t, %lf\n",np[i],dis[i],gam[i],hm[i],tm[i]);
    	}


    	hipFree(d_x);
    	hipFree(d_y);
    	hipFree(d_z);
    	hipFree(d_np);
    	hipFree(d_dis);
    	hipFree(d_gam);
    	hipFree(d_hm);
    	hipFree(d_tm);
    	hipFree(d_uvxazm);
    	hipFree(d_uvyazm);
    	hipFree(d_uvzdec);
    	hipFree(d_uvhdec);
    	hipFree(d_csatol);
    	hipFree(d_csdtol);
    	hipFree(d_bandwh);
    	hipFree(d_bandwd);
    	hipFree(d_atol);
    	hipFree(d_vr);
    	hipFree(d_ivtype);
    	hipFree(d_ivtail);
    	hipFree(d_ivhead);
    	free(h_np);
    	free(h_dis);
    	free(h_gam);
    	free(h_hm);
    	free(h_tm);
	return 0;
//end routine
}







/*

TODO:

- Add documentation in each routine
- Add diagrams (UML?) of the sequence and interactions between the CPU and GPU
- Document the memory optimization proposed.

*/








